#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda_types.h"
#include "../defs.h"
//#include "cuda_global.h"
//#include "cuda_functions.h"
//#include "cuda_routines.h"

#define _PROFILE

//#define THREAD 1024 // 2048 for A100
//#define BLOCK 32    // 32 for A100 

//#define THREAD 128 // 2048 for A100
#define THREAD 1 // 2048 for A100
#define BLOCK 1024    // 32 for A100 

#define _six 6
#define _two 2
#define _seven 7

#define ESP2 1e-4
#define new_size(A) ((A > 1024) ? int(pow(2,ceil(log(A)/log(2.0)))) : 1024)


static int NNB;
//static int NumNeighborMax;
static REAL time_send, time_grav, time_out, time_nb;
static long long numInter;
static int icall,ini,isend;
static int nbodymax;
static int devid, numGPU;
static bool is_open = false;
static bool devinit = false;
static bool first   = true;
static int variable_size;
//const int memory_size = 512;
//BackgroundParticle *h_background, *d_background;

extern REAL *h_ptcl, *d_ptcl; //, *background;
extern REAL *h_result, *d_result;
extern REAL *d_diff, *d_magnitudes, *d_r2;
extern int *h_neighbor, *d_neighbor, *h_num_neighbor, *d_num_neighbor;
extern int *d_target;

REAL *h_ptcl=nullptr, *d_ptcl=nullptr;; //, *background;
REAL *h_result=nullptr, *d_result=nullptr;
REAL *d_diff=nullptr,*d_magnitudes=nullptr, *d_r2=nullptr;
int *h_neighbor=nullptr, *d_neighbor=nullptr, *d_num_neighbor=nullptr, *h_num_neighbor=nullptr;
int *d_target=nullptr;


extern hipStream_t stream;
hipStream_t stream;

extern REAL *h_diff, *h_magnitudes;
REAL *h_diff, *h_magnitudes;

/*************************************************************************
 *	 Computing Acceleration
 *************************************************************************/
__global__	void initialize(REAL* result, int* neighbor, int* num_neighbor, REAL* diff, REAL *magnitudes, int n, int m, int* subset);
__global__ void compute_pairwise_diff_subset(const REAL* ptcl, REAL* diff, int n, int m, const int* subset);
__global__ void compute_magnitudes_subset(const REAL *r2, const REAL* diff, REAL* magnitudes, int n, int m, int* subset);
__global__ void compute_forces_subset(const REAL* ptcl, REAL *diff, const REAL* magnitudes, int n, int m, const int* subset);
__global__ void assign_neighbor(int *neighbor, int* num_neighbor, const REAL* r2, const REAL* magnitudes, int n, int m, const int *subset);
__global__ void reduce_forces(const REAL *diff, REAL *result, int n, int m);

__device__ void _addition(Result &result, const Result res);
__device__ void _copy(Result &result, const Result res);


// CUDA kernel to compute the forces for a subset of particles
__global__ void print_forces_subset(REAL* result, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < m) {
		printf("acc: (%d) = %e\n", idx, result[_six*idx]);
				/*
				atomicAdd(&result[i+1], scale * diff[six_idx + 1]);
				atomicAdd(&result[i+2], scale * diff[six_idx + 2]);

				atomicAdd(&result[i+3], scale * (diff[six_idx + 3] - magnitudes[idx+1]*diff[six_idx    ]/magnitudes[idx]));
				atomicAdd(&result[i+4], scale * (diff[six_idx + 4] - magnitudes[idx+1]*diff[six_idx + 1]/magnitudes[idx]));
				atomicAdd(&result[i+5], scale * (diff[six_idx + 5] - magnitudes[idx+1]*diff[six_idx + 2]/magnitudes[idx]));
				*/
	}
}

void initializeCudaAndCublas(hipblasHandle_t* handle);

void GetAcceleration(
		int NumTarget,
		int h_target_list[],
		REAL acc[][3],
		REAL adot[][3],
		int NumNeighbor[],
		int **NeighborList
		) {

	assert(is_open);
	assert((NumTarget > 0) && (NumTarget <= NNB));

	hipError_t cudaStatus;
	hipError_t error;

	//hipStreamCreate(&stream);

	hipblasHandle_t handle;
	initializeCudaAndCublas(&handle);

	/*
	for(int i=0; i<NumTarget; i++) {
		d_result[i].clear();
		d_neighbor[i].clear();
		d_dist = 0.;
	}
	*/
	/*
	fprintf(stderr,"\ntargets=");
	for(int i=0; i<NumTarget; i++) {
		fprintf(stderr,"%d, ", h_target_list[i]);
	}
	fprintf(stderr,"\n");
	*/


	//toDevice(h_target_list, d_target, NumTarget, stream);
	toDevice(h_target_list, d_target, NumTarget);

	// Kernel launch parameters
	dim3 blockSize(variable_size);
	dim3 gridSize(NumTarget);
	//dim3 gridSize((NumTarget * NNB + blockSize.x - 1) / blockSize.x);

	// Compute pairwise differences for the subset
	initialize<<<gridSize, blockSize, 0, stream>>>\
		(d_result, d_neighbor, d_num_neighbor, d_diff, d_magnitudes, NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	compute_pairwise_diff_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_ptcl, d_diff, NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	compute_magnitudes_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_r2, d_diff, d_magnitudes, NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	// Compute gravitational forces for the subset
	compute_forces_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_ptcl, d_diff, d_magnitudes, NNB, NumTarget, d_target);
	assign_neighbor<<<gridSize, blockSize, 0, stream>>>\
		(d_neighbor, d_num_neighbor, d_r2, d_magnitudes, NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	reduce_forces<<<gridSize, blockSize, 0, stream>>>\
		(d_diff, d_result, NNB, NumTarget);
	hipDeviceSynchronize();

	/*
	print_forces_subset<<<gridSize, blockSize>>>\
		(d_result, NumTarget);
		*/




	/*
	toHost(h_diff, d_diff, _six*NumTarget*NNB);
	for (int i = 0; i < NumTarget; ++i) {
		//std::cerr << "PID=" << h_target_list[i] << std::endl;
		for (int j = 0; j < NNB; ++j) {
			std::cerr << h_diff[_six*(i * NNB + j)] << " ";
		}
		std::cerr << std::endl;
	}
	*/

	//toHost(h_result  , d_result  , variable_size, stream);
	//toHost(h_neighbor, d_neighbor, variable_size, stream);


	hipStreamSynchronize(stream); // Wait for all operations to finish

	toHost(h_result      , d_result      ,           _six*NumTarget);
	toHost(h_neighbor    , d_neighbor    , NumNeighborMax*NumTarget);
	toHost(h_num_neighbor, d_num_neighbor,                NumTarget);
	//printf("CUDA: transfer to host done\n");


	//hipStreamSynchronize(stream); // Wait for all operations to finish


	for (int i=0;i<NumTarget;i++) {
		for (int j=0;j<h_num_neighbor[i];j++) {
			NeighborList[i][j] = h_neighbor[NumNeighborMax*i+j];
			//fprintf(stderr, "%d, ", NeighborList[i][j]);
		}
		//fprintf(stdout, "%d (%d) neighbors of %d", h_target_list[i], i, h_num_neighbor[i]);
		/*
		for (int j=0;j<h_num_neighbor[i];j++) {
			fprintf(stdout, "%d, ", NeighborList[i][j]);
		}
		*/
		//fprintf(stdout, "\n");
		NumNeighbor[i] = h_num_neighbor[i];

		/*
		fprintf(stderr, "PID=%d: a=(%.4e,%.4e,%.4e), adot=(%.4e,%.4e,%.4e)\n",
				h_target_list[i],
				h_result[_six*i],
				h_result[_six*i+1],
				h_result[_six*i+2],
				h_result[_six*i+3],
				h_result[_six*i+4],
				h_result[_six*i+5]
				);
				*/
	}

	// out data
	for (int i=0; i<NumTarget; i++) {
		acc[i][0]  = h_result[_six*i];
		acc[i][1]  = h_result[_six*i+1];
		acc[i][2]  = h_result[_six*i+2];
		adot[i][0] = h_result[_six*i+3];
		adot[i][1] = h_result[_six*i+4];
		adot[i][2] = h_result[_six*i+5];
	}

	hipblasDestroy(handle);
	/*
	my_free(h_background , d_background);
	my_free(h_result     , d_result);
	my_free(h_target     , d_target);
	my_free(h_neighbor   , d_neighbor);
	*/
	//hipStreamDestroy(stream);
	//my_free_d(do_neighbor);
	//printf("CUDA: done?\n");
}


__global__	void initialize(REAL* result, int* neighbor, int* num_neighbor, REAL* diff, REAL *magnitudes, int n, int m, int* subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	diff[_six*idx    ] = 0.;
	diff[_six*idx + 1] = 0.;
	diff[_six*idx + 2] = 0.;
	diff[_six*idx + 3] = 0.;
	diff[_six*idx + 4] = 0.;
	diff[_six*idx + 5] = 0.;

	magnitudes[_two*idx    ] = 0.;
	magnitudes[_two*idx + 1] = 0.;

	if (idx < m * n) {
		int i = idx / n;
		int j = idx % n;

		if (j == 0) {
			result[_six*i] = 0.;
			result[_six*i + 1] = 0.;
			result[_six*i + 2] = 0.;
			result[_six*i + 3] = 0.;
			result[_six*i + 4] = 0.;
			result[_six*i + 5] = 0.;
			num_neighbor[i] = 0;
			/*
			for (j=0; j<NumNeighborMax; j++)
				neighbor[NumNeighborMax*i+j] = 0;
				*/
		}
	}
}

// CUDA kernel to compute pairwise differences for a subset of particles
__global__ void compute_pairwise_diff_subset(const REAL* ptcl, REAL* diff, int n, int m, const int* subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < m * n) {
		int i = subset[idx / n];
		int j = idx % n;
		idx *= _six;
		i *= _seven;
		j *= _seven;

		diff[idx]   = ptcl[j]   - ptcl[i];
		diff[idx+1] = ptcl[j+1] - ptcl[i+1];
		diff[idx+2] = ptcl[j+2] - ptcl[i+2];
		diff[idx+3] = ptcl[j+3] - ptcl[i+3];
		diff[idx+4] = ptcl[j+4] - ptcl[i+4];
		diff[idx+5] = ptcl[j+5] - ptcl[i+5];

		//printf("(%d,%d) = %e, %e, %e\n", i/_seven, j/_seven,  ptcl[i], ptcl[j], diff[idx]);
	}
}


__global__ void compute_magnitudes_subset(const REAL *r2, const REAL* diff, REAL* magnitudes, int n, int m, int* subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n * m) {
		int i = subset[idx / n];
		int j = idx % n;
		int six_idx = _six*idx;
		int two_idx = _two*idx;


		magnitudes[two_idx]   += diff[(six_idx)]    *diff[(six_idx)];
		magnitudes[two_idx]   += diff[(six_idx) + 1]*diff[(six_idx) + 1];
		magnitudes[two_idx]   += diff[(six_idx) + 2]*diff[(six_idx) + 2];
		magnitudes[two_idx+1] += diff[(six_idx)]    *diff[(six_idx) + 3];
		magnitudes[two_idx+1] += diff[(six_idx) + 1]*diff[(six_idx) + 4];
		magnitudes[two_idx+1] += diff[(six_idx) + 2]*diff[(six_idx) + 5];

		//printf("(%d,%d) = %e, %e\n", i, j,  magnitudes[two_idx], r2[i]);

		if (magnitudes[two_idx] <= r2[i]) {
			//printf("(%d, %d): %e, %e\n",subset[i], j, magnitudes[two_idx], r2[i]);
			magnitudes[two_idx]   = -magnitudes[two_idx];
		}
	}
}


// CUDA kernel to compute the forces for a subset of particles
__global__ void compute_forces_subset(const REAL* ptcl, REAL *diff, const REAL* magnitudes, int n, int m, const int* subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < m * n) {
		//int i = subset[idx / n];
		int i = idx / n;
		int j = idx % n;
		int six_idx = idx*_six;
		idx *= _two;
		REAL acc[Dim], adot[Dim];

		if (magnitudes[idx] <= 0.) {
			acc[0]  = 0.;
			acc[1]  = 0.;
			acc[2]  = 0.;
			adot[0] = 0.;
			adot[1] = 0.;
			adot[2] = 0.;
		}
		else {
			REAL scale = ptcl[_seven*j+6] / (magnitudes[idx] * sqrtf(magnitudes[idx]));
			acc[0]  = scale * diff[six_idx];
			acc[1]  = scale * diff[six_idx + 1];
			acc[2]  = scale * diff[six_idx + 2];

			adot[0] = scale * (diff[six_idx + 3] - 3*magnitudes[idx+1]*diff[six_idx    ]/magnitudes[idx]);
			adot[1] = scale * (diff[six_idx + 4] - 3*magnitudes[idx+1]*diff[six_idx + 1]/magnitudes[idx]);
			adot[2] = scale * (diff[six_idx + 5] - 3*magnitudes[idx+1]*diff[six_idx + 2]/magnitudes[idx]);
		}

		diff[six_idx]   = acc[0];
		diff[six_idx+1] = acc[1];
		diff[six_idx+2] = acc[2];
		diff[six_idx+3] = adot[0];
		diff[six_idx+4] = adot[1];
		diff[six_idx+5] = adot[2];

		//printf("compute_forces: (%d, %d) = %e\n", i, j,  diff[six_idx]);
	}
}


/*
__device__ REAL warpReduce(REAL val) {
	val += __shfl_down_sync(0xffffffff, val, 16);
	val += __shfl_down_sync(0xffffffff, val, 8);
	val += __shfl_down_sync(0xffffffff, val, 4);
	val += __shfl_down_sync(0xffffffff, val, 2);
	val += __shfl_down_sync(0xffffffff, val, 1);
	return val;
}
*/

__inline__ __device__ REAL warpReduce(REAL val)
{
	for (int offset = warpSize/2; offset > 0; offset /= 2) 
		val += __shfl_down_sync(0xffffffff, val, offset);
	return val;
}


__global__ void reduce_forces(const REAL *diff, REAL *result, int n, int m) {
	int idx = blockIdx.x * n + threadIdx.x;
	__shared__ REAL warpSum[64]; // Assumes max 32 warps per block
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;
	REAL sum;
	int i = blockIdx.x;
	int j = threadIdx.x;
	int six_idx = _six*(i*n+j);
	int k;
	

#pragma unroll 
	for (k=0;k<_six;k++) {
		sum = (i < m && j < n) ? diff[six_idx+k] : 0;
		/*
		if (k == 0)
			if (i < m && j < n)
				printf("(%d,%d) = %e\n", blockIdx.x,threadIdx.x, diff[six_idx+k]);
				*/

		// Warp reduce
		sum = warpReduce(sum);

		// Block reduce
		if (lane == 0) warpSum[wid] = sum;
		__syncthreads();

		if (wid == 0)
		{
			sum = (threadIdx.x < blockDim.x / warpSize) ? warpSum[lane] : 0;
			sum = warpReduce(sum);
			if (lane == 0) result[_six*i+k] = sum;
		}
	}
}



__global__ void assign_neighbor(int *neighbor, int* num_neighbor, const REAL* r2, const REAL* magnitudes, int n, int m, const int *subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < m) {
		int i = subset[idx];
		int k = 0;

		for (int j = 0; j < n; j++) {
			if (i != j) {
				k = _two*(n*idx+j);
				if (magnitudes[k] < 0) {
					//printf("(%d, %d,%d) = %d, %e, %e\n", idx, i, j, num_neighbor[idx], magnitudes[k], r2[i]);
					neighbor[NumNeighborMax*idx+num_neighbor[idx]] = j;
					num_neighbor[idx]++;
					if (num_neighbor[idx] > 100)  {
						//printf("Error: (%d, %d,%d) = %d, %e, %e\n", idx, i, j, num_neighbor[idx], magnitudes[k], r2[i]);
						assert(num_neighbor[idx] < 100);
						return;
					}
				}
			}
		}
	}
}




/*
__global__ void compute_forces_subset(REAL* result, const REAL* ptcl, const REAL *diff, const REAL* magnitudes, int n, int m, const int* subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//if (idx < m * n) {
	int i = subset[idx / n];
	int j = idx % n;
	int six_idx = idx*_six;
	REAL scale;
	idx *= _two;
	__shared__ REAL res[_six];

	if (threadIdx.x == 0) { 
		res[0]=0;
		res[1]=0;
		res[2]=0;
		res[3]=0;
		res[4]=0;
		res[5]=0;
	}

	if (idx >= m * n || i == j || magnitudes[idx] <= 0.) {
		atomicAdd(&res[0], 0.);
		atomicAdd(&res[1], 0.);
		atomicAdd(&res[2], 0.);

		atomicAdd(&res[3], 0.);
		atomicAdd(&res[4], 0.);
		atomicAdd(&res[5], 0.);
	}
	else  {
		scale = ptcl[_seven*j+6] / (magnitudes[idx] *sqrtf(magnitudes[idx]));
		i *= _six;
		atomicAdd(&res[0], scale * diff[six_idx]);
		atomicAdd(&res[1], scale * diff[six_idx + 1]);
		atomicAdd(&res[2], scale * diff[six_idx + 2]);

		atomicAdd(&res[3], scale * (diff[six_idx + 3] - magnitudes[idx+1]*diff[six_idx    ]/magnitudes[idx]));
		atomicAdd(&res[4], scale * (diff[six_idx + 4] - magnitudes[idx+1]*diff[six_idx + 1]/magnitudes[idx]));
		atomicAdd(&res[5], scale * (diff[six_idx + 5] - magnitudes[idx+1]*diff[six_idx + 2]/magnitudes[idx]));
	}
	__syncthreads();

	if (threadIdx.x == 0) { 
		result[i]   = res[0];
		result[i+1] = res[1];
		result[i+2] = res[2];
		result[i+3] = res[3];
		result[i+4] = res[4];
		result[i+5] = res[5];
	}
}
*/


void initializeCudaAndCublas(hipblasHandle_t* handle) {
	hipError_t cudaStat = hipSetDevice(0);
	if (cudaStat != hipSuccess) {
		std::cerr << "hipSetDevice failed!" << std::endl;
		exit(1);
	}

	hipblasStatus_t stat = hipblasCreate(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		std::cerr << "CUBLAS initialization failed!" << std::endl;
		exit(1);
	}
}


/*************************************************************************
 *	 Communication with HOST
 *************************************************************************/
void _ReceiveFromHost(
		int _NNB,
		REAL m[],
		REAL x[][3],
		REAL v[][3],
		REAL r2[],
		REAL mdot[]
		){
	//time_send -= get_wtime();
	nbodymax       = 100000000;
	NNB            = _NNB;
	//NumNeighborMax = _NumNeighborMax;
	isend++;
	assert(NNB <= nbodymax);
	hipError_t cudaStatus;

	printf("CUDA: receive starts\n");
	//my_allocate(&h_background, &d_background_tmp, new_size(NNB));
	//hipMemcpyToSymbol(HIP_SYMBOL(d_background), &d_background_tmp, new_size(NNB)*sizeof(BackgroundParticle));
	
	if ((first) || (new_size(NNB) > variable_size )) {
		variable_size = new_size(NNB);
		if (!first) {
			my_free(h_ptcl				 , d_ptcl);
			my_free(h_result       , d_result);
			my_free(h_neighbor     , d_neighbor);
			my_free(h_num_neighbor , d_num_neighbor);
			hipFree(d_target);
			hipFree(d_r2);
			hipFree(d_diff);
			hipFree(d_magnitudes);
		}
		else {
			first = false;
		}
		my_allocate(&h_ptcl         , &d_ptcl        ,         _seven*variable_size); // x,v,m
		my_allocate(&h_result       , &d_result      ,           _six*variable_size);
		my_allocate(&h_num_neighbor , &d_num_neighbor,                variable_size);
		my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax*variable_size);
		hipMalloc((void**)&d_r2        ,        variable_size * sizeof(REAL));
		hipMalloc((void**)&d_target    ,        variable_size * sizeof(int));
		hipMalloc((void**)&d_diff      , _six * variable_size * variable_size * sizeof(REAL));
		hipMalloc((void**)&d_magnitudes, _two * variable_size * variable_size * sizeof(REAL));
		//hipHostMalloc((void**)&h_diff          , _six * variable_size * variable_size * sizeof(REAL));
		//hipHostMalloc((void**)&h_magnitudes    , _two * variable_size * variable_size * sizeof(REAL));
	}


	for (int j=0; j<NNB; j++) {
		for (int dim=0; dim<Dim; dim++) {
			h_ptcl[_seven*j+dim]   = x[j][dim];
			h_ptcl[_seven*j+dim+3] = v[j][dim];
		}
		h_ptcl[_seven*j+6] = m[j];
		//h_particle[j].setParticle(m[j], x[j], v[j], r2[j], mdot[j]);
	}

	//toDevice(h_background,d_background,variable_size);
	toDevice(h_ptcl,d_ptcl, _seven*NNB, stream);
	toDevice(r2    ,d_r2  ,        NNB, stream);
	//fprintf(stdout, "CUDA: receive done\n");
}



void _InitializeDevice(int irank){

	std::cout << "Initializing CUDA ..." << std::endl;
	// Select CUDA device (optional)
	int device = 0; // Choose GPU device 0
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	hipStreamCreate(&stream);

	std::cout << "There are " << deviceCount << " GPUs." << std::endl;
	if (device < 0 || device >= deviceCount) {
		    // Handle invalid device index
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");


	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	

	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);


	hipSetDevice(device);

	// Initialize CUDA context
	/*
	hipError_t cudaStatus = hipFree(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
		return;
	}
	*/

	is_open = true;
	// CUDA is now initialized and ready to be used
	std::cout << "CUDA initialized successfully!" << std::endl;

	/*
	if(devinit) return;

	hipGetDeviceCount(&numGPU);
	assert(numGPU > 0);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list)
	{
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		if (p) {
			devid = atoi(p);
			numGPU++;
		}
		assert(numGPU > 0);
	}else{
		devid=irank%numGPU;
	}
	hipSetDevice(devid);

#ifdef PROFILE
	//  if(!irank)fprintf(stderr, "***********************\n");
	//  if(!irank)fprintf(stderr, "Initializing NBODY6/GPU library\n");
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");
	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);
	//  if(!irank)fprintf(stderr, "***********************\n");
#endif
	devinit = true;
	*/
}



void _OpenDevice(const int irank){
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend = 0;

	//select GPU========================================//
	_InitializeDevice(irank);

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;


#ifdef PROFILE
	//	fprintf(stderr, "RANK: %d ******************\n",irank);
	//	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "# Open GPU regular force - rank: %d\n", irank);
	//fprintf(stderr, "***********************\n");
#endif
}



void _CloseDevice() {
	if(!is_open) {
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;


	hipError_t error;

	printf("CUDA: ?!! ...\n");
	//my_free(&h_result    , &d_result);
	fprintf(stderr, "result ...\n");
	//my_free(&h_target    , &d_target);
	fprintf(stderr, "target ...\n");
	//my_free(&h_neighbor  , &d_neighbor);
	fprintf(stderr, "neighbor ...\n");
	//my_free(&h_background, &d_background);

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		// Handle error
	}

#ifdef PROFILE
	fprintf(stderr, "Closed NBODY6/GPU library\n");
	fprintf(stderr, "rank: %d***************\n",devid);
	fprintf(stderr, "time send : %f sec\n", time_send);
	fprintf(stderr, "time grav : %f sec\n", time_grav);
	fprintf(stderr, "time nb   : %f sec\n", time_nb);
	fprintf(stderr, "time out  : %f sec\n", time_out);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
}



void _ProfileDevice(int irank) {
#ifdef PROFILE
	if(icall) {
		fprintf(stderr,"[R.%d-D.%d GPU Reg.F ] Nsend %d  Ngrav %d  <Ni> %d   send(s) %f grav(s) %f  nb(s) %f  out(s) %f  Perf.(Gflops) %f\n",irank,devid,isend,icall,ini/isend,time_send,time_grav,time_nb,time_out,60.e-9*numInter/time_grav);
	}
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend= 0;
#else
	return;
#endif
}


#define mexPrintf printf

inline void gpuMemReport(size_t * avail, size_t * total, 
		        const char * title = 0, const size_t * free = 0, const bool sense = true) 
{
	char tstring[32] = { '\0' };
	hipMemGetInfo(avail, total);  

	if (free) {
		if (title) {
			strncpy(tstring, title, 31);
		}
		mexPrintf("%s Memory avaliable: Free: %zu, Total: %zu, %s: %zu\n",
				tstring, *avail, *total, (sense) ? "Allocated\0" : "Freed\0", 
				(sense) ? (*free - *avail) : (*avail - *free));
	} else {
		mexPrintf("Memory avaliable: Free: %zu, Total: %zu\n", *avail, *total);  
	}
}



extern "C" {
	void InitializeDevice(int *irank){
		_InitializeDevice(*irank);
	}
	void OpenDevice(const int *irank){
		_OpenDevice(*irank);
	}
	void CloseDevice(){
		_CloseDevice();
	}
	void SendToDevice(int *_NNB, REAL m[], REAL x[][3], REAL v[][3], REAL r2[], REAL mdot[]) {
		_ReceiveFromHost(*_NNB, m, x, v, r2, mdot);
	}
	void ProfileDevice(int *irank){
		_ProfileDevice(*irank);
	}
	void CalculateAccelerationOnDevice(int *NumTarget, int *h_target_list, REAL acc[][3], REAL adot[][3], int NumNeighbor[], int **NeighborList) {
		GetAcceleration(*NumTarget, h_target_list, acc, adot, NumNeighbor, NeighborList);
	}
}

