#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../defs.h"
#include "cuda_defs.h"
#include "cuda_kernels.h"
#include "cuda_routines.h"

#ifdef NSIGHT
#include <nvToolsExt.h>
#endif

#ifdef THRUST
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/find.h>
#endif


static int NNB;
static CUDA_REAL time_send, time_grav, time_out, time_nb;
static long long numInter;
static int icall,ini,isend;
static int nbodymax;
static int devid, numGPU;
static bool is_open = false;
static bool devinit = false;
static bool first   = true;
static int variable_size;


extern CUDA_REAL *h_ptcl, *d_ptcl; //, *background;
extern CUDA_REAL *h_result, *d_result;
extern CUDA_REAL *d_diff, *d_magnitudes, *d_r2;
extern int *d_target;

CUDA_REAL *h_ptcl=nullptr, *d_ptcl=nullptr;; //, *background;
CUDA_REAL *h_result=nullptr, *d_result=nullptr;
CUDA_REAL *d_diff=nullptr,*d_magnitudes=nullptr, *d_r2=nullptr;
int *d_target=nullptr;

#define TEST_CUBLAS
#ifndef TEST_CUBLAS
extern int *h_neighbor, *d_neighbor, *h_num_neighbor, *d_num_neighbor;
int *h_neighbor=nullptr, *d_neighbor=nullptr, *d_num_neighbor=nullptr, *h_num_neighbor=nullptr;

#else
extern bool *h_neighbor, *d_neighbor;
extern int *h_num_neighbor;
bool *h_neighbor=nullptr, *d_neighbor=nullptr;
int *h_num_neighbor=nullptr; // added by wispedia
#endif

extern hipStream_t stream;
hipStream_t stream;

extern CUDA_REAL *h_diff, *h_magnitudes;
CUDA_REAL *h_diff, *h_magnitudes;


void reduce_forces_cublas(hipblasHandle_t handle, const CUDA_REAL *diff, CUDA_REAL *result, int n, int m) {

	CUDA_REAL *d_matrix;
    hipMalloc(&d_matrix, m * n * sizeof(CUDA_REAL));

    // Create a vector of ones for the summation
    double *ones;
    hipMalloc(&ones, n * sizeof(double));
    double *h_ones = new double[n];
    for (int i = 0; i < n; ++i) {
        h_ones[i] = 1.0;
    }
    hipMemcpy(ones, h_ones, n * sizeof(double), hipMemcpyHostToDevice);
    // Initialize result array to zero
    hipMemset(result, 0, m * 6 * sizeof(double));

    const double alpha = 1.0;
    const double beta = 0.0;

    // Sum over the second axis (n) for each of the 6 elements
    for (int i = 0; i < _six; ++i) {

		hipblasDcopy(handle, m * n, diff + i, _six, d_matrix, 1);
        hipblasDgemv(
            handle,
            HIPBLAS_OP_T,  // Transpose
            n,            // Number of rows of the matrix A
            m,            // Number of columns of the matrix A
            &alpha,       // Scalar alpha
            d_matrix, // Pointer to the first element of the i-th sub-matrix
            n,     // Leading dimension of the sub-matrix
            ones,         // Pointer to the vector x
            1,            // Increment between elements of x
            &beta,        // Scalar beta
            result + i, // Pointer to the first element of the result vector
            _six             // Increment between elements of the result vector
        );
    }
    // Cleanup
    delete[] h_ones;
    hipFree(ones);
	hipFree(d_matrix);
}

#ifdef THRUST

struct less_than_zero
{
    __host__ __device__ bool operator()(const float x) const
    {
        return x < 0;
    }
};


void reduce_forces_thrust(const CUDA_REAL *diff, CUDA_REAL *result, int n, int m) {
    // Wrap raw pointers with Thrust device pointers
    thrust::device_ptr<const CUDA_REAL> d_diff(diff);
    thrust::device_ptr<CUDA_REAL> d_result(result);

    // Initialize result array to zero
    thrust::fill(d_result, d_result + m * 6, 0);

    // Sum over the second axis (n) for each of the 6 elements
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < m; ++j) {
            // Calculate the start and end pointers for the current sub-matrix
            thrust::device_ptr<const CUDA_REAL> start = d_diff + i + j * n * 6;
            thrust::device_ptr<const CUDA_REAL> end = start + n * 6;

            // Create a thrust device vector from start to end
            thrust::device_vector<CUDA_REAL> sub_matrix(start, end);

            // Reduce the sub-matrix and store the result
            d_result[i + j * 6] = thrust::reduce(sub_matrix.begin(), sub_matrix.end());
        }
    }
}


void reduce_neighbors(hipblasHandle_t handle, int *neighbor, int* num_neighbor, CUDA_REAL *magnitudes, int n, int m, int* subset) {

	CUDA_REAL *d_matrix;
    hipMalloc(&d_matrix, m * n * sizeof(CUDA_REAL));
    hipblasDcopy(handle, m * n, magnitudes, _two, d_matrix, 1);


	for (int row = 0; row < m; ++row){
		CUDA_REAL val = 1.0;
        hipMemcpy(d_matrix + row * n + subset[row], &val, sizeof(CUDA_REAL), hipMemcpyHostToDevice);
	}

    // Wrap raw device pointers with thrust device pointers
    thrust::device_ptr<const CUDA_REAL> d_ptr(d_matrix);
    thrust::device_ptr<int> d_neighbor(neighbor);
    thrust::device_ptr<int> d_num_neighbor(num_neighbor);

    // Process each row
    for (int row = 0; row < m; ++row) {
        auto row_start = d_ptr + row * n;
        auto row_end = row_start + n;

        thrust::counting_iterator<int> index_sequence(0);

        // Use thrust::copy_if to select indices where elements are less than zero
        auto end = thrust::copy_if(index_sequence, index_sequence + n, row_start, d_neighbor + row * NumNeighborMax, less_than_zero());

        // Calculate the number of negative elements in the current row
        int num_neg_elements = thrust::distance(d_neighbor + row * NumNeighborMax, end);

        if (num_neg_elements > NumNeighborMax) {
            hipFree(d_matrix);
            throw std::runtime_error("Number of negative elements exceeds NumNeighborMax");
        }

        d_num_neighbor[row] = num_neg_elements;
    }

    hipFree(d_matrix);
}
#endif
/*************************************************************************
 *	 Computing Acceleration
 *************************************************************************/

void GetAcceleration(
		int NumTarget,
		int h_target_list[],
		CUDA_REAL acc[][3],
		CUDA_REAL adot[][3],
		int NumNeighbor[],
		int **NeighborList
		) {

	assert(is_open);
	assert((NumTarget > 0) && (NumTarget <= NNB));

	int minGridSize, blockSize, gridSize;
	int sharedMemSize;

	//hipStreamCreate(&stream);

	hipblasHandle_t handle;
	initializeCudaAndCublas(&handle);

	/*
	for(int i=0; i<NumTarget; i++) {
		d_result[i].clear();
		d_neighbor[i].clear();
		d_dist = 0.;
	}
	*/
	/*
	fprintf(stderr,"\ntargets=");
	for(int i=0; i<NumTarget; i++) {
		fprintf(stderr,"%d, ", h_target_list[i]);
	}
	fprintf(stderr,"\n");
	*/


	//toDevice(h_target_list, d_target, NumTarget, stream);
	toDevice(h_target_list, d_target, NumTarget, stream);

	// Kernel launch parameters
	//dim3 blockSize(variable_size);
	//dim3 gridSize(NumTarget);
	//dim3 gridSize((NumTarget * NNB + blockSize.x - 1) / blockSize.x);

	// Compute pairwise differences for the subset

	//blockSize = variable_size;
	//gridSize = NumTarget;
	int total_data_num = new_size(NNB*NumTarget);
	/******* Initialize *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	initialize, 0, 0));	
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	initialize<<<gridSize, blockSize, 0, stream>>>\
		(d_result, d_diff, d_magnitudes, NNB, NumTarget, d_target);
	hipDeviceSynchronize();


	/******* Differencese *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	compute_pairwise_diff_subset, 0, 0));	
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	compute_pairwise_diff_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_ptcl, d_diff, NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	/******* Magnitudes *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	compute_magnitudes_subset, 0, 0));	
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	compute_magnitudes_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_r2, d_diff, d_magnitudes, NNB, NumTarget, d_target, d_neighbor); // changed by wispedia
	hipDeviceSynchronize();

	/******* Force *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	compute_forces_subset, 0, 0));
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	compute_forces_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_ptcl, d_diff, d_magnitudes, NNB, NumTarget, d_target);




	#ifndef TEST_CUBLAS
	/******* Neighborhood *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
				assign_neighbor, 0, 0));
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	
	//blockSize = std::min(blockSize, 512);
	//gridSize = (NNB * NumTarget + blockSize - 1) / blockSize;

	//blockSize = variable_size;
	//gridSize = NumTarget;

	#define MAX_SIZE 9
	sharedMemSize = ((MAX_SIZE+1)*blockSize) * sizeof(int);
	assign_neighbor<<<gridSize, blockSize, sharedMemSize, stream>>>\
		(d_neighbor, d_num_neighbor, d_r2, d_magnitudes, NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	/******* Reduction *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	reduce_forces, 0, 0));
	gridSize = (total_data_num + blockSize - 1) / blockSize;
	//blockSize = NNB;
	//gridSize  = NumTarget;
	//blockSize = 128;
	//blockSize = variable_size;
	//gridSize = NumTarget;


	//	sharedMemSize = 256 * sizeof(double);
	reduce_forces<<<gridSize, blockSize, 0, stream>>>\
		(d_diff, d_result, NNB, NumTarget);
	hipDeviceSynchronize();
	//print_forces_subset<<<gridSize, blockSize>>>\
		(d_result, NumTarget);
	#else
	/******* Neighborhood (new) *********/
	// reduce_neighbors(handle, d_neighbor, d_num_neighbor, d_magnitudes, NNB, NumTarget, h_target_list);
	hipDeviceSynchronize();

	#ifdef NSIGHT
    nvtxRangePushA("Reduction");
	#endif
	/******* Reduction *********/
	reduce_forces_cublas(handle, d_diff, d_result, NNB, NumTarget); //test by wispedia
	//reduce_forces_thrust(d_diff, d_result, NNB, NumTarget);
	hipDeviceSynchronize();

	#ifdef NSIGHT
	nvtxRangePop();
	#endif
	//print_forces_subset<<<gridSize, blockSize>>>\
		(d_result, NumTarget);	
	#endif
	/*
	toHost(h_diff, d_diff, _six*NumTarget*NNB);
	for (int i = 0; i < NumTarget; ++i) {
		//std::cerr << "PID=" << h_target_list[i] << std::endl;
		for (int j = 0; j < NNB; ++j) {
			std::cerr << h_diff[_six*(i * NNB + j)] << " ";
		}
		std::cerr << std::endl;
	}
	*/

	//toHost(h_result  , d_result  , variable_size, stream);
	//toHost(h_neighbor, d_neighbor, variable_size, stream);


	hipStreamSynchronize(stream); // Wait for all operations to finish

	toHost(h_result      , d_result      ,           _six*NumTarget);

	#ifdef TEST_CUBLAS
	#ifdef NSIGHT
    nvtxRangePushA("Neighbor in CPU");
	#endif

	toHost(h_neighbor, d_neighbor, NNB * NumTarget);
	for (int i=0;i<NumTarget;i++) {
		int k = 0;
	    int* targetNeighborList = NeighborList[i]; // Cache the row pointer
	    int target = h_target_list[i]; // Cache the target value

		for (int j=0;j<NNB;j++) {
			if (h_neighbor[i * NNB + j] && (target != j)) {
				if (k<NumNeighborMax){
					targetNeighborList[k] = j;
					}
				k++;
			}
		}
		NumNeighbor[i] = k; // h_num_neighbor[i];
	}
	#ifdef NSIGHT
	nvtxRangePop();
	#endif
	
	#else
	toHost(h_neighbor    , d_neighbor    , NumNeighborMax*NumTarget);
	toHost(h_num_neighbor, d_num_neighbor,                NumTarget);

	//printf("CUDA: transfer to host done\n");


	//hipStreamSynchronize(stream); // Wait for all operations to finish

	for (int i=0;i<NumTarget;i++) {
		for (int j=0;j<h_num_neighbor[i];j++) {
			NeighborList[i][j] = h_neighbor[NumNeighborMax*i+j];
		}
		NumNeighbor[i] = h_num_neighbor[i];

		/*
		fprintf(stderr, "%d (%d) neighbors of %d = ", i, h_target_list[i], h_num_neighbor[i]);
		for (int j=0;j<h_num_neighbor[i];j++) {
			fprintf(stderr, "%d, ", NeighborList[i][j]);
		}
		fprintf(stderr, "\n");
		*/

		/*
		fprintf(stderr, "PID=%d: a=(%.4e,%.4e,%.4e), adot=(%.4e,%.4e,%.4e)\n",
				h_target_list[i],
				h_result[_six*i],
				h_result[_six*i+1],
				h_result[_six*i+2],
				h_result[_six*i+3],
				h_result[_six*i+4],
				h_result[_six*i+5]
				);
				*/
	}
	#endif

	//fprintf(stderr, "\n");
	#ifdef NSIGHT
	nvtxRangePushA("Move h_result to acc and adot");
	#endif
	// out data
	for (int i=0; i<NumTarget; i++) {
		acc[i][0]  = h_result[_six*i];
		acc[i][1]  = h_result[_six*i+1];
		acc[i][2]  = h_result[_six*i+2];
		adot[i][0] = h_result[_six*i+3];
		adot[i][1] = h_result[_six*i+4];
		adot[i][2] = h_result[_six*i+5];
	}
	#ifdef NSIGHT
	nvtxRangePop();
	#endif

	hipblasDestroy(handle);
	/*
	my_free(h_background , d_background);
	my_free(h_result     , d_result);
	my_free(h_target     , d_target);
	my_free(h_neighbor   , d_neighbor);
	*/
	//hipStreamDestroy(stream);
	//my_free_d(do_neighbor);
	//printf("CUDA: done?\n");
}







/*************************************************************************
 *	 Communication with HOST
 *************************************************************************/
void _ReceiveFromHost(
		int _NNB,
		CUDA_REAL m[],
		CUDA_REAL x[][3],
		CUDA_REAL v[][3],
		CUDA_REAL r2[],
		CUDA_REAL mdot[]
		){
	//time_send -= get_wtime();
	nbodymax       = 100000000;
	NNB            = _NNB;
	//NumNeighborMax = _NumNeighborMax;
	isend++;
	assert(NNB <= nbodymax);
	hipError_t cudaStatus;

	//printf("CUDA: receive starts\n");
	//my_allocate(&h_background, &d_background_tmp, new_size(NNB));
	//hipMemcpyToSymbol(HIP_SYMBOL(d_background), &d_background_tmp, new_size(NNB)*sizeof(BackgroundParticle));
	
	if ((first) || (new_size(NNB) > variable_size )) {
		variable_size = new_size(NNB);
		if (!first) {
			my_free(h_ptcl				 , d_ptcl);
			my_free(h_result       , d_result);
			my_free(h_neighbor     , d_neighbor);
			// my_free(h_num_neighbor , d_num_neighbor);
			hipHostFree(h_num_neighbor);
			hipFree(d_target);
			hipFree(d_r2);
			hipFree(d_diff);
			hipFree(d_magnitudes);

		}
		else {
			first = false;
		}
		my_allocate(&h_ptcl         , &d_ptcl        ,         _seven*variable_size); // x,v,m
		my_allocate(&h_result       , &d_result      ,           _six*variable_size);
		// my_allocate(&h_num_neighbor , &d_num_neighbor,                variable_size);
		// my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax*variable_size);
		my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax*variable_size);
		hipMalloc((void**)&d_r2        ,        variable_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_target    ,        variable_size * sizeof(int));
		hipMalloc((void**)&d_diff      , _six * variable_size * variable_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_magnitudes, _two * variable_size * variable_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_diff          , _six * variable_size * variable_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_magnitudes    , _two * variable_size * variable_size * sizeof(CUDA_REAL));
		#ifdef TEST_CUBLAS
		my_allocate(&h_neighbor     , &d_neighbor    , variable_size * variable_size);
		hipHostMalloc((void**)&h_num_neighbor, variable_size * sizeof(int));
		#endif
		
	}


	for (int j=0; j<NNB; j++) {
		for (int dim=0; dim<Dim; dim++) {
			h_ptcl[_seven*j+dim]   = x[j][dim];
			h_ptcl[_seven*j+dim+3] = v[j][dim];
		}
		h_ptcl[_seven*j+6] = m[j];
		//h_particle[j].setParticle(m[j], x[j], v[j], r2[j], mdot[j]);
	}

	//toDevice(h_background,d_background,variable_size);
	toDevice(h_ptcl,d_ptcl, _seven*NNB, stream);
	toDevice(r2    ,d_r2  ,        NNB, stream);
	//fprintf(stdout, "CUDA: receive done\n");
}



void _InitializeDevice(int irank){

	std::cout << "Initializing CUDA ..." << std::endl;
	// Select CUDA device (optional)
	int device = 0; // Choose GPU device 0
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	hipStreamCreate(&stream);

	std::cout << "There are " << deviceCount << " GPUs." << std::endl;
	if (device < 0 || device >= deviceCount) {
		    // Handle invalid device index
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");


	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	

	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);


	hipSetDevice(device);

	// Initialize CUDA context
	/*
	hipError_t cudaStatus = hipFree(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
		return;
	}
	*/

	is_open = true;
	// CUDA is now initialized and ready to be used
	std::cout << "CUDA initialized successfully!" << std::endl;

	/*
	if(devinit) return;

	hipGetDeviceCount(&numGPU);
	assert(numGPU > 0);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list)
	{
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		if (p) {
			devid = atoi(p);
			numGPU++;
		}
		assert(numGPU > 0);
	}else{
		devid=irank%numGPU;
	}
	hipSetDevice(devid);

#ifdef PROFILE
	//  if(!irank)fprintf(stderr, "***********************\n");
	//  if(!irank)fprintf(stderr, "Initializing NBODY6/GPU library\n");
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");
	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);
	//  if(!irank)fprintf(stderr, "***********************\n");
#endif
	devinit = true;
	*/
}



void _OpenDevice(const int irank){
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend = 0;

	//select GPU========================================//
	_InitializeDevice(irank);

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;


#ifdef PROFILE
	//	fprintf(stderr, "RANK: %d ******************\n",irank);
	//	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "# Open GPU regular force - rank: %d\n", irank);
	//fprintf(stderr, "***********************\n");
#endif
}



void _CloseDevice() {
	if(!is_open) {
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;


	hipError_t error;

	printf("CUDA: ?!! ...\n");
	//my_free(&h_result    , &d_result);
	fprintf(stderr, "result ...\n");
	//my_free(&h_target    , &d_target);
	fprintf(stderr, "target ...\n");
	//my_free(&h_neighbor  , &d_neighbor);
	fprintf(stderr, "neighbor ...\n");
	//my_free(&h_background, &d_background);

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		// Handle error
	}

#ifdef PROFILE
	fprintf(stderr, "Closed NBODY6/GPU library\n");
	fprintf(stderr, "rank: %d***************\n",devid);
	fprintf(stderr, "time send : %f sec\n", time_send);
	fprintf(stderr, "time grav : %f sec\n", time_grav);
	fprintf(stderr, "time nb   : %f sec\n", time_nb);
	fprintf(stderr, "time out  : %f sec\n", time_out);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
}



void _ProfileDevice(int irank) {
#ifdef PROFILE
	if(icall) {
		fprintf(stderr,"[R.%d-D.%d GPU Reg.F ] Nsend %d  Ngrav %d  <Ni> %d   send(s) %f grav(s) %f  nb(s) %f  out(s) %f  Perf.(Gflops) %f\n",irank,devid,isend,icall,ini/isend,time_send,time_grav,time_nb,time_out,60.e-9*numInter/time_grav);
	}
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend= 0;
#else
	return;
#endif
}


#define mexPrintf printf

inline void gpuMemReport(size_t * avail, size_t * total, 
		        const char * title = 0, const size_t * free = 0, const bool sense = true) 
{
	char tstring[32] = { '\0' };
	hipMemGetInfo(avail, total);  

	if (free) {
		if (title) {
			strncpy(tstring, title, 31);
		}
		mexPrintf("%s Memory avaliable: Free: %zu, Total: %zu, %s: %zu\n",
				tstring, *avail, *total, (sense) ? "Allocated\0" : "Freed\0", 
				(sense) ? (*free - *avail) : (*avail - *free));
	} else {
		mexPrintf("Memory avaliable: Free: %zu, Total: %zu\n", *avail, *total);  
	}
}



extern "C" {
	void InitializeDevice(int *irank){
		_InitializeDevice(*irank);
	}
	void OpenDevice(const int *irank){
		_OpenDevice(*irank);
	}
	void CloseDevice(){
		_CloseDevice();
	}
	void SendToDevice(int *_NNB, CUDA_REAL m[], CUDA_REAL x[][3], CUDA_REAL v[][3], CUDA_REAL r2[], CUDA_REAL mdot[]) {
		_ReceiveFromHost(*_NNB, m, x, v, r2, mdot);
	}
	void ProfileDevice(int *irank){
		_ProfileDevice(*irank);
	}
	void CalculateAccelerationOnDevice(int *NumTarget, int *h_target_list, CUDA_REAL acc[][3], CUDA_REAL adot[][3], int NumNeighbor[], int **NeighborList) {
		GetAcceleration(*NumTarget, h_target_list, acc, adot, NumNeighbor, NeighborList);
	}
}

